#include "hip/hip_runtime.h"
//Vector functions
#include <iostream>

#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/fill.h>
#include <thrust/inner_product.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/execution_policy.h>
#include <thrust/system_error.h>
#include <vector_functions.h>


float L2sq_distance(const thrust::device_vector<float> &a, const thrust::device_vector<float> &b)
{

  thrust::plus<float> binary_op;
  thrust::device_vector<float> r_vec;
  r_vec.resize(a.size());

  //compute the squared euclidean distance between the a and the empty space description vector
  
  thrust::transform(a.begin(), a.end(), b.begin(), r_vec.begin(), sq_diff<float>());
  return thrust::reduce(r_vec.begin(), r_vec.end(), 0.0f, binary_op);
  
}

void print_device_vector(thrust::device_vector<float> &d_vec)
{
  thrust::for_each(thrust::device, d_vec.begin(), d_vec.end(), printf_functor());
}


float L1_distance(thrust::device_vector<float> &a, thrust::device_vector<float> &b)
{

  thrust::plus<float> binary_op;
  thrust::device_vector<float> r_vec;
  r_vec.resize(a.size());
  
  thrust::transform(a.begin(), a.end(), b.begin(), r_vec.begin(), abs_diff<float>());
  return thrust::reduce(r_vec.begin(), r_vec.end(), 0.0f, binary_op); 
}

float em_distance(thrust::device_vector<float> &a, thrust::device_vector<float> &b)
{
/*
  //normalizing input vectors
  thrust::device_vector<float> a_normalized = a;
  thrust::device_vector<float> b_normalized = b;
  float norm_a = std::sqrt(thrust::inner_product(a.begin(), a.end(),a.begin(),0.0f));
  float norm_b = std::sqrt(thrust::inner_product(b.begin(), b.end(),b.begin(),0.0f));
  thrust::transform(a_normalized.begin(), a_normalized.end(), a_normalized.begin(), thrust::placeholders::_1 /= norm_a);
  thrust::transform(b_normalized.begin(), b_normalized.end(), b_normalized.begin(), thrust::placeholders::_1 /= norm_b);
*/
  //computing the cdf's
  thrust::device_vector<float> a_cdf = a/*_normalized*/;
  thrust::device_vector<float> b_cdf = b/*_normalized*/;
  thrust::inclusive_scan(a_cdf.begin(), a_cdf.end(),a_cdf.begin());
  thrust::inclusive_scan(b_cdf.begin(), b_cdf.end(),b_cdf.begin());

  return L1_distance(a_cdf,b_cdf);

}

float line_search_combination_parallel(
const  thrust::device_vector<float> &v1, 
const  thrust::device_vector<float> &v2, 
const  thrust::device_vector<float> &reference,
size_t numel,  
  thrust::device_vector<float> &combination )
{
  thrust::device_vector<float> w1;               w1.resize(numel);
  thrust::device_vector<float> w2;               w2.resize(numel);
  thrust::device_vector<float> weighted_v1;      weighted_v1.resize(numel);
  thrust::device_vector<float> weighted_v2;      weighted_v2.resize(numel);
  combination.resize(numel);

  float best_error = std::numeric_limits<float>::infinity();
  float best_weight = 0.5;
  for (int w_i = 0; w_i < 101; ++w_i)
  {
    float w_f = float(w_i)/100.f;
    thrust::fill(w1.begin(), w1.end(), w_f);
    thrust::fill(w2.begin(), w2.end(), 1.0f - w_f);

    thrust::transform(w1.begin(), w1.end(), v1.begin(), weighted_v1.begin(),
      thrust::multiplies<float>());
    thrust::transform(w2.begin(), w2.end(), v2.begin(), weighted_v2.begin(),
      thrust::multiplies<float>());
    thrust::transform(weighted_v1.begin(), weighted_v1.end(), weighted_v2.begin(), combination.begin(),
      thrust::plus<float>());
   
    float current_err = L2sq_distance(combination, reference);
    if (current_err < best_error)
    {
      best_error = current_err;
      best_weight = w_f;
    }
  }
  
  thrust::fill(w1.begin(), w1.end(), best_weight);
  thrust::fill(w2.begin(), w2.end(), 1.0f - best_weight);

  thrust::transform(w1.begin(), w1.end(), v1.begin(), weighted_v1.begin(),
    thrust::multiplies<float>());
  thrust::transform(w2.begin(), w2.end(), v2.begin(), weighted_v2.begin(),
    thrust::multiplies<float>());
  thrust::transform(weighted_v1.begin(), weighted_v1.end(), weighted_v2.begin(), combination.begin(),
    thrust::plus<float>());

  return best_weight;
}


float line_search_combination_sequential(
const  thrust::device_vector<float> &v1, 
const  thrust::device_vector<float> &v2, 
const  thrust::device_vector<float> &reference,
size_t numel,  
  thrust::device_vector<float> &combination )
{
  thrust::device_vector<float> w1;               w1.resize(numel);
  thrust::device_vector<float> weighted_v2;      weighted_v2.resize(numel);
  combination.resize(numel);

  float best_error = std::numeric_limits<float>::infinity();
  float best_weight = 0.5;
  for (int w_i = 0; w_i < 101; ++w_i)
  {
    float w_f = float(w_i)/100.f;
    thrust::fill(w1.begin(), w1.end(), w_f);

    thrust::transform(w1.begin(), w1.end(), v2.begin(), weighted_v2.begin(),
      thrust::multiplies<float>());
    thrust::transform(v1.begin(), v1.end(), weighted_v2.begin(), combination.begin(),
      thrust::plus<float>());
   
    float current_err = L2sq_distance(combination, reference);
    if (current_err < best_error)
    {
      best_error = current_err;
      best_weight = w_f;
    }
  }
  
  thrust::fill(w1.begin(), w1.end(), best_weight);

  thrust::transform(w1.begin(), w1.end(), v2.begin(), weighted_v2.begin(),
    thrust::multiplies<float>());
  thrust::transform(v1.begin(), v1.end(), weighted_v2.begin(), combination.begin(),
    thrust::plus<float>());

  return best_weight;
}

void weighted_vector_sum_parallel(  
    const  thrust::device_vector<float> &v1, 
    const  thrust::device_vector<float> &v2, 
    const  float weight,
    size_t numel,  
    thrust::device_vector<float> &combination )
{
  thrust::device_vector<float> w1;               w1.resize(numel);
  thrust::device_vector<float> w2;               w2.resize(numel);
  thrust::device_vector<float> weighted_v1;      weighted_v1.resize(numel);
  thrust::device_vector<float> weighted_v2;      weighted_v2.resize(numel);
  combination.resize(numel);

  thrust::fill(w1.begin(), w1.end(), weight);
  thrust::fill(w2.begin(), w2.end(), 1.0f - weight);

  thrust::transform(w1.begin(), w1.end(), v1.begin(), weighted_v1.begin(),
    thrust::multiplies<float>());
  thrust::transform(w2.begin(), w2.end(), v2.begin(), weighted_v2.begin(),
    thrust::multiplies<float>());
  thrust::transform(weighted_v1.begin(), weighted_v1.end(), weighted_v2.begin(), combination.begin(),
    thrust::plus<float>());
}

void weighted_vector_sum_sequential(  
    const  thrust::device_vector<float> &v1, 
    const  thrust::device_vector<float> &v2,
    const float weight, 
    size_t numel,  
    thrust::device_vector<float> &combination )
{
  thrust::device_vector<float> w;               w.resize(numel);
  thrust::device_vector<float> weighted_v2;      weighted_v2.resize(numel);
  combination.resize(numel);
  
  thrust::fill(w.begin(), w.end(), weight);

  thrust::transform(w.begin(), w.end(), v2.begin(), weighted_v2.begin(),
    thrust::multiplies<float>());
  thrust::transform(v1.begin(), v1.end(), weighted_v2.begin(), combination.begin(),
    thrust::plus<float>());
}


void vector_difference(const thrust::device_vector<float> &a, const thrust::device_vector<float> &b, thrust::device_vector<float> &c)
{
  c.resize(a.size());
  thrust::transform(a.begin(), a.end(), b.begin(), c.begin(),
      thrust::minus<float>());
}

void test_vector_functions(void)
{
  thrust::device_vector<float> A;
  thrust::device_vector<float> B;
  thrust::device_vector<float> R;

  A.push_back(0.2f);
  A.push_back(0.4f);
  A.push_back(0.6f);
  A.push_back(0.8f);
  A.push_back(1.0f);

  B.push_back(-0.1f);
  B.push_back(0.2f);
  B.push_back(-0.3f);
  B.push_back(0.4f);
  B.push_back(-0.5f);

  R.push_back(0.3f);
  R.push_back(0.4f);
  R.push_back(0.5f);
  R.push_back(0.6f);
  R.push_back(0.7f);



  std::cout << "testing vector functions..." << std::endl;
  std::cout << "A : " << std::endl;
  print_device_vector(A);
  std::cout << std::endl;
  hipDeviceSynchronize();
  
  std::cout << "B : " << std::endl;
  print_device_vector(B);
  std::cout << std::endl;
  hipDeviceSynchronize();
  
  std::cout << "R : " << std::endl;
  print_device_vector(R);
  std::cout << std::endl;
  hipDeviceSynchronize();
  
  std::cout << "L2sq_distance(A,A): "<< L2sq_distance(A,A) << std::endl;
  std::cout << "L1_distance(A,A): "<< L1_distance(A,A) << std::endl;
  hipDeviceSynchronize();
  std::cout << "L2sq_distance(A,B): "<< L2sq_distance(A,B) << std::endl;
  std::cout << "L1_distance(A,B): "<< L1_distance(A,B) << std::endl;
  hipDeviceSynchronize();
  std::cout << "L2sq_distance(B,A): "<< L2sq_distance(B,A) << std::endl;
  std::cout << "L1_distance(B,A): "<< L1_distance(B,A) << std::endl;
  hipDeviceSynchronize();
  std::cout << "L2sq_distance(B,B): "<< L2sq_distance(B,B) << std::endl;
  std::cout << "L1_distance(B,B): "<< L1_distance(B,B) << std::endl;
  hipDeviceSynchronize();
  std::cout << "L2sq_distance(A,R): "<< L2sq_distance(A,R) << std::endl;
  std::cout << "L1_distance(A,R): "<< L1_distance(A,R) << std::endl;
  hipDeviceSynchronize();
  std::cout << "L2sq_distance(B,R): "<< L2sq_distance(B,R) << std::endl;
  std::cout << "L1_distance(B,R): "<< L1_distance(B,R) << std::endl;
  hipDeviceSynchronize();
  std::cout << "L2sq_distance(R,R): "<< L2sq_distance(R,R) << std::endl;
  std::cout << "L1_distance(R,R): "<< L1_distance(R,R) << std::endl;
  hipDeviceSynchronize();
  std::cout << "L2sq_distance(R,A): "<< L2sq_distance(R,A) << std::endl;
  std::cout << "L1_distance(R,A): "<< L1_distance(R,A) << std::endl;
  hipDeviceSynchronize();
  std::cout << "L2sq_distance(R,B): "<< L2sq_distance(R,B) << std::endl;
  std::cout << "L1_distance(R,B): "<< L1_distance(R,B) << std::endl;
  hipDeviceSynchronize();
  std::cout << "L2sq_distance(R,R): "<< L2sq_distance(R,R) << std::endl;
  std::cout << "L1_distance(R,R): "<< L1_distance(R,R) << std::endl;
  hipDeviceSynchronize();
  
  std::cout << "vector_difference(A,B,C)" << std::endl;
  thrust::device_vector<float> C;
  vector_difference(A,B,C);
  std::cout << "A : " << std::endl;
  print_device_vector(A);
  std::cout << std::endl;
  hipDeviceSynchronize();
  
  std::cout << "B : " << std::endl;
  print_device_vector(B);
  std::cout << std::endl;
  hipDeviceSynchronize();
  
  std::cout << "C : " << std::endl;
  print_device_vector(C);
  std::cout << std::endl;
  hipDeviceSynchronize();


  std::cout << "weighted_vector_sum_sequential(A, B, 0.25, 5, C)" << std::endl;
  weighted_vector_sum_sequential(A, B, 0.25, 5, C);
  std::cout << "A : " << std::endl;
  print_device_vector(A);
  std::cout << std::endl;
  hipDeviceSynchronize();
  
  std::cout << "B : " << std::endl;
  print_device_vector(B);
  std::cout << std::endl;
  hipDeviceSynchronize();
  
  std::cout << "C : " << std::endl;
  print_device_vector(C);
  std::cout << std::endl;
  hipDeviceSynchronize();
  
  std::cout << "weighted_vector_sum_parallel(A, B, 0.25, 5, C)" << std::endl;
  weighted_vector_sum_parallel(A, B, 0.25, 5, C);
  std::cout << "A : " << std::endl;
  print_device_vector(A);
  std::cout << std::endl;
  hipDeviceSynchronize();
  
  std::cout << "B : " << std::endl;
  print_device_vector(B);
  std::cout << std::endl;
  hipDeviceSynchronize();
  
  std::cout << "C : " << std::endl;
  print_device_vector(C);
  std::cout << std::endl;
  hipDeviceSynchronize();
  
  float weight = 0;

  std::cout << "line_search_combination_sequential(A, B, R, 5, C)" << std::endl;
  weight = line_search_combination_sequential(A, B, R, 5, C);
  
  std::cout << "w : " << weight << std::endl;

  std::cout << "A : " << std::endl;
  print_device_vector(A);
  std::cout << std::endl;
  hipDeviceSynchronize();
  
  std::cout << "B : " << std::endl;
  print_device_vector(B);
  std::cout << std::endl;
  hipDeviceSynchronize();
  
  std::cout << "C : " << std::endl;
  print_device_vector(C);
  std::cout << std::endl;
  hipDeviceSynchronize();
  
  weight = 0;
  std::cout << "line_search_combination_parallel(A, B, R, 5, C)" << std::endl;
  weight = line_search_combination_parallel(A, B, R, 5, C);
  
  std::cout << "w : " << weight << std::endl;

  std::cout << "A : " << std::endl;
  print_device_vector(A);
  std::cout << std::endl;
  hipDeviceSynchronize();
  
  std::cout << "B : " << std::endl;
  print_device_vector(B);
  std::cout << std::endl;
  hipDeviceSynchronize();
  
  std::cout << "C : " << std::endl;
  print_device_vector(C);
  std::cout << std::endl;
  hipDeviceSynchronize();


}