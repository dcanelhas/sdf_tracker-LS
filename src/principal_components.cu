#define OC_NEW_STYLE_INCLUDES 1
#include "chooseser.h"
#include "principal_components.h"
#include "vector_functions.h"


#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/execution_policy.h>
#include <thrust/system_error.h>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <string>
#include <iostream>
#include <vector>
#include <ctime>


void principal_components::load_dictionary(std::string &filename)
{
    input_dim = TSDF_BLOCKSIZE*TSDF_BLOCKSIZE*TSDF_BLOCKSIZE;
    // sprintf(weight_file, "sparse_pca_128.pickle");

    std::cout<< "Reading file: " << filename << "." <<std::endl;

    Array<real_4> w_arr;
    Val w_val; LoadValFromFile( filename.c_str() , w_val, SERIALIZE_P2); w_arr = (w_val);

//    host_weights[layer_number][ nodes[layer_number]*n_out + n_in ];

    std::cout<< "Got " << w_arr.length() <<  " weights." <<std::endl;
    reduced_dim = w_arr.length()/input_dim;

    for(int w_idx = 0; w_idx<w_arr.length(); ++w_idx )
    host_weights.push_back((w_arr[w_idx]));

    dev_weights = host_weights;

    thrust::device_vector<float> empty_data;
    empty_data.resize(input_dim);
    thrust::fill(empty_data.begin(), empty_data.end(),1.0f);
    encode(empty_data, word_for_empty);

    std::cout<< "loaded weights " << w_arr.length() <<  " weights." <<std::endl;

}

void principal_components::load_mean(std::string &filename)
{
    input_dim = TSDF_BLOCKSIZE*TSDF_BLOCKSIZE*TSDF_BLOCKSIZE;
    // sprintf(weight_file, "sparse_pca_128.pickle");

    std::cout<< "Reading file: " << filename << "." <<std::endl;

    Array<real_4> w_arr;
    Val w_val; LoadValFromFile( filename.c_str() , w_val, SERIALIZE_P2); w_arr = (w_val);

//    host_weights[layer_number][ nodes[layer_number]*n_out + n_in ];

    std::cout<< "Got " << w_arr.length() <<  " weights." <<std::endl;

    for(int w_idx = 0; w_idx<w_arr.length(); ++w_idx )
    host_mean.push_back((w_arr[w_idx]));

    dev_mean = host_mean;


}


void principal_components::encode(thrust::device_vector<float> &input, thrust::device_vector<float> &output)
{

  // hipblasSgemv performs the computation y = alpha*( A*x ) + beta * y
  // since y is our bias vector, we need to make a copy so it doesn't get replaced with the
  // results of the computation. A represents the weights and x is the input to the current layer
  // After each such computation, we also have to evaluate the activation function of the network
  // This is done using thrust::for_each()

  thrust::device_vector<float> input_minus_mean;
  input_minus_mean.resize(input_dim);
  output.resize(reduced_dim);

  //find the mean
//  float mean = thrust::reduce(thrust::device, input.begin(), input.end())/float(input_dim);
//  thrust::fill(output.begin(), output.end(), mean);

  //make an input-sized vector containing the mean
  // mean_vector.resize(input_dim);
  // thrust::fill(mean_vector.begin(), mean_vector.end(), mean);

  //subtract the mean from the input to make it centered
  thrust::transform(input.begin(), input.end(), dev_mean.begin(), input_minus_mean.begin(), subtract<float>() );

  float alf = 1.0f; float beta = 0;

  //hipStream_t Stream;
  //hipStreamCreate(&Stream);

  hipblasHandle_t handle;
  (hipblasCreate(&handle));

  //hipblasSetStream(handle, Stream);

  (hipblasSgemv( handle, HIPBLAS_OP_T, 4096, reduced_dim, &alf, (thrust::raw_pointer_cast(&dev_weights[0])), 4096,
    thrust::raw_pointer_cast(&input_minus_mean[0]), 1, &beta,thrust::raw_pointer_cast(&output[0]),1));

  (hipblasDestroy(handle));
  //hipStreamDestroy(Stream);


}


bool principal_components::describes_empty(thrust::device_vector<float> &input, const float threshold)
{
  thrust::plus<float> binary_op;
  thrust::device_vector<float> r_vec;
  r_vec.resize(reduced_dim);

  thrust::transform(input.begin(), input.end(), word_for_empty.begin(), r_vec.begin(), abs_diff<float>());
  float distance = thrust::reduce(r_vec.begin(), r_vec.end(), 0.0f, binary_op);

  return (distance < threshold) ? true : false;
}


void principal_components::decode(thrust::device_vector<float> &input, thrust::device_vector<float> &output)
{
  // hipblasSgemv performs the computation y = alpha*( A*x ) + beta * y

  // output.resize(4096);
  output = dev_mean;
  // thrust::fill(output.begin(), output.end(),1.0f);
  float alf = 1.0f;
  //beta holds the mean that was extracted earlier, during encoding. it has to be added to the final result
  float beta = 1.0f;//input[reduced_dim];

  //hipStream_t Stream;
  //hipStreamCreate(&Stream);

  hipblasHandle_t handle;
  (hipblasCreate(&handle));

  //hipblasSetStream(handle, Stream);

  (hipblasSgemv( handle, HIPBLAS_OP_N, 4096, reduced_dim, &alf, (thrust::raw_pointer_cast(&dev_weights[0])), 4096,
    thrust::raw_pointer_cast(&input[0]), 1, &beta,thrust::raw_pointer_cast(&output[0]),1));

  (hipblasDestroy(handle));
  //hipStreamDestroy(Stream);
}

void principal_components::compare(thrust::host_vector<float> &input, thrust::host_vector<float> &output)
{
  thrust::device_vector<float> original = input;
  thrust::device_vector<float> reconstructed = original;
  thrust::device_vector<float> descriptor;

  encode(original,descriptor);

//  std::cout << "encoding original " << original.size() << " elements to " << descriptor.size() << " elements." << std::endl;


  decode(descriptor,reconstructed);
//  std::cout << "decoding descriptor " << descriptor.size() << " elements to " << reconstructed.size() << " elements." << std::endl;
  output = reconstructed;
}


void principal_components::compare_gold(thrust::host_vector<float> &input ,thrust::host_vector<float> &output )
{
/*
  int nodes[num_layers+1] =     { 4096,     2048,     1024,     512 ,     256,      128,      256,      512,      1024,     2048,     4096};
  int num_outputs[num_layers] = {           nodes[1], nodes[2], nodes[3], nodes[4], nodes[5], nodes[6], nodes[7], nodes[8], nodes[9], nodes[10] };
  int num_inputs[num_layers] =  { nodes[0], nodes[1], nodes[2], nodes[3], nodes[4], nodes[5], nodes[6], nodes[7], nodes[8], nodes[9]};

  output.resize(nodes[num_layers]);

  //allocating neural network container
  float* outputs[num_layers];
  float* activations[num_layers];

  for (int layer_number = 0; layer_number < num_layers; ++layer_number)
  {
    outputs[layer_number] = new float[num_outputs[layer_number]];
    activations[layer_number] = new float[num_outputs[layer_number]];
  }

  //set activations and output to zero so we can compute the sums
  for(uint layer_number = 0; layer_number < num_layers; layer_number++)
    for (uint fill = 0; fill < num_outputs[layer_number]; ++fill)
      outputs[layer_number][fill] = activations[layer_number][fill] = 0.f;

  for(uint layer_number = 0; layer_number < num_layers; ++layer_number)
  {
    // #pragma omp parallel
    for(uint n_out = 0; n_out < num_outputs[layer_number]; ++n_out)
    {
      //compute activations layerwise
      for(uint n_in = 0; n_in < num_inputs[layer_number]; ++n_in)
      {

        activations[layer_number][n_out] += (layer_number == 0) ? (input[n_in]) * host_weights[layer_number][ nodes[layer_number]*n_out + n_in ] :
                                            outputs[layer_number-1][n_in] * host_weights[layer_number][ nodes[layer_number]*n_out + n_in ];
      }
      outputs[layer_number][n_out] = logistic(activations[layer_number][n_out] + host_biases[layer_number][n_out] );
    }
  }

  for(int d=0; d < input_dim; ++d)
  {
    output[d] = outputs[num_layers-1][d];
  }

  for (int del = 0; del < num_layers; ++del)
  {
    if(activations[del]!=NULL)
      delete[] activations[del];
    if(outputs[del]!=NULL)
      delete[] outputs[del];
  }*/
}
