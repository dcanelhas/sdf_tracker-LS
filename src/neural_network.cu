#include "hip/hip_runtime.h"
#define OC_NEW_STYLE_INCLUDES 1
#include "chooseser.h"
#include "neural_network.h"
#include "vector_functions.h"

#include <thrust/fill.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/execution_policy.h>
#include <thrust/system_error.h>

#include <math.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <string>
#include <iostream>
#include <vector>
#include <ctime>


struct tanh_functor
{
  __host__ __device__
  void operator()(float &x)
  {
    x = tanh(x);
  }
};


inline float logistic(float x){ return 1.0f/(1.0f + expf(-x)); }
struct logistic_functor
{
  __host__ __device__
  void operator()(float &x)
  {
    x = 1.0f/(1.0f + expf(-x));
  }
};

void neural_network::load_network(const std::string &directory, int layers)
{  
  
  num_layers = layers;
  network_structure.push_back(TSDF_BLOCKSIZE*TSDF_BLOCKSIZE*TSDF_BLOCKSIZE);

  for (size_t i = 0; i < num_layers; ++i)
  {

    char str[15];
    sprintf(str, "%d", i+1);

    std::string weight_file = directory + "fw" + str + ".pickle" ;
    std::string bias_file = directory + "fb" + str + ".pickle" ;
    // char bias_file[50];
    // sprintf(weight_file, directory.c_str() + "fw%d.pickle", i+1);
    // sprintf(bias_file, "directory.c_str() + fb%d.pickle", i+1);

    std::cout<< "Reading files: " << weight_file << ", " << bias_file << "." <<std::endl;

    Array<real_4> w_arr;
    Array<real_4> b_arr;
    Val w_val; LoadValFromFile( weight_file.c_str() , w_val, SERIALIZE_P2); w_arr = (w_val);
    Val b_val; LoadValFromFile( bias_file.c_str(), b_val, SERIALIZE_P2); b_arr = (b_val);

    network_structure.push_back(b_arr.length());

//    host_weights[layer_number][ nodes[layer_number]*n_out + n_in ];

    std::cout<< "Got " << w_arr.length() <<  " weights and "<< b_arr.length() << " biases." <<std::endl;

    for(int w_idx = 0; w_idx<w_arr.length(); ++w_idx )
      host_weights[i].push_back((w_arr[w_idx]));

    for(int b_idx = 0; b_idx<b_arr.length(); ++b_idx )
      host_biases[i].push_back((b_arr[b_idx]));

    dev_biases[i] = host_biases[i];  dev_weights[i] = host_weights[i];
  }
  
  thrust::device_vector<float> empty_data;
  empty_data.resize(TSDF_BLOCKSIZE*TSDF_BLOCKSIZE*TSDF_BLOCKSIZE, 1.0f);
  encode(empty_data, word_for_empty);
  reduced_dim = word_for_empty.size();
  std::cout<<"Done."<<std::endl;
}

void neural_network::encode(thrust::device_vector<float> &input, thrust::device_vector<float> &output)
{
  // hipblasSgemv performs the computation y = alpha*( A*x ) + beta * y
  // since y is our bias vector, we need to make a copy so it doesn't get replaced with the
  // results of the computation. A represents the weights and x is the input to the current layer
  // After each such computation, we also have to evaluate the activation function of the network
  // This is done using thrust::for_each()
  
  for (int L = 0; L < num_layers/2-1; ++L)
  {
    dev_results[L] = dev_biases[L];
  }
  output = dev_biases[num_layers/2-1];
  float alf = 1.0f; float beta = 1.0f;
  
  //hipStream_t Stream;
  //hipStreamCreate(&Stream);  

  hipblasHandle_t handle;
  (hipblasCreate(&handle));

  //hipblasSetStream(handle, Stream);
 
  hipblasSgemv( 
                handle, 
                HIPBLAS_OP_T, 
                network_structure[0], 
                network_structure[1], 
                &alf, 
                (thrust::raw_pointer_cast(dev_weights[0].data())), 
                network_structure[0],
                thrust::raw_pointer_cast(&input[0]), 
                1, 
                &beta,
                thrust::raw_pointer_cast(dev_results[0].data()),
                1
              );
  thrust::for_each(thrust::device,dev_results[0].begin(),dev_results[0].end(),logistic_functor());
  
  for (int i = 1; i < num_layers/2-1; ++i)
  {
    hipblasSgemv(  
                  handle, 
                  HIPBLAS_OP_T, 
                  network_structure[i], 
                  network_structure[i+1], 
                  &alf, 
                  thrust::raw_pointer_cast(dev_weights[i].data()), 
                  network_structure[i],
                  thrust::raw_pointer_cast(dev_results[i-1].data()), 
                  1, 
                  &beta,
                  thrust::raw_pointer_cast(dev_results[i].data()),
                  1
                );
    thrust::for_each(thrust::device,dev_results[i].begin(),dev_results[i].end(),logistic_functor());
  }

  hipblasSgemv( 
    handle, 
    HIPBLAS_OP_T, 
    network_structure[num_layers/2-1], 
    network_structure[num_layers/2], 
    &alf, 
    (thrust::raw_pointer_cast(dev_weights[num_layers/2-1].data())), 
    network_structure[num_layers/2-1],
    thrust::raw_pointer_cast(dev_results[num_layers/2-2].data()), 
    1, 
    &beta,
    thrust::raw_pointer_cast(&output[0]),
    1
  );
  thrust::for_each(thrust::device,output.begin(),output.end(),logistic_functor());
 
  (hipblasDestroy(handle));
  //hipStreamDestroy(Stream);  

}

bool neural_network::describes_empty(thrust::device_vector<float> &input, float threshold)
{
  thrust::plus<float> binary_op;
  thrust::device_vector<float> r_vec;
  r_vec.resize(reduced_dim);

  thrust::transform(input.begin(), input.end(), word_for_empty.begin(), r_vec.begin(), sq_diff<float>());
  float distance = thrust::reduce(r_vec.begin(), r_vec.end(), 0.0f, binary_op);

  return (distance < threshold) ? true : false;

}

void neural_network::decode(thrust::device_vector<float> &input, thrust::device_vector<float> &output, activation_fcn which_fcn)
{

  // hipblasSgemv performs the computation y = alpha*( A*x ) + beta * y
  // since y is our bias vector, we need to make a copy so it doesn't get replaced with the
  // results of the computation. A represents the weights and x is the input to the current layer
  // After each such computation, we also have to evaluate the activation function of the network
  // This is done using thrust::for_each()


  for (int L = num_layers/2; L < num_layers-1; ++L)
  {
    dev_results[L] = dev_biases[L];
  }
  output = dev_biases[num_layers-1];
  
  float alf = 1.0f; float beta = 1.0f;
  //hipStream_t Stream;
  //hipStreamCreate(&Stream);  

  hipblasHandle_t handle;
  (hipblasCreate(&handle));

  //hipblasSetStream(handle, Stream);

  
  hipblasSgemv( 
                  handle, 
                  HIPBLAS_OP_T, 
                  network_structure[num_layers/2], 
                  network_structure[num_layers/2+1], 
                  &alf, 
                  thrust::raw_pointer_cast(dev_weights[num_layers/2].data()), 
                  network_structure[num_layers/2],
                  thrust::raw_pointer_cast(&input[0]), 
                  1, 
                  &beta,thrust::raw_pointer_cast(dev_results[num_layers/2].data()),
                  1
                );
  if(which_fcn == SIGMOID)
    thrust::for_each(thrust::device,dev_results[num_layers/2].begin(),dev_results[num_layers/2].end(),logistic_functor());
  else if(which_fcn == TANH)
    thrust::for_each(thrust::device,dev_results[num_layers/2].begin(),dev_results[num_layers/2].end(), tanh_functor());


  for (int i = num_layers/2+1; i < num_layers-1; ++i)
  {
    hipblasSgemv( 
                  handle, 
                  HIPBLAS_OP_T, 
                  network_structure[i], 
                  network_structure[i+1], 
                  &alf, 
                  thrust::raw_pointer_cast(dev_weights[i].data()), 
                  network_structure[i],
                  thrust::raw_pointer_cast(dev_results[i-1].data()), 
                  1, 
                  &beta,thrust::raw_pointer_cast(dev_results[i].data()),
                  1
                );
  if(which_fcn == SIGMOID)
      thrust::for_each(thrust::device,dev_results[i].begin(),dev_results[i].end(),logistic_functor());
  else if(which_fcn == TANH)
      thrust::for_each(thrust::device,dev_results[i].begin(),dev_results[i].end(),tanh_functor());

  }
  hipblasSgemv( 
              handle, 
              HIPBLAS_OP_T, 
              network_structure[num_layers-1], 
              network_structure[num_layers], 
              &alf, 
              thrust::raw_pointer_cast(dev_weights[num_layers-1].data()), 
              network_structure[num_layers-1],
              thrust::raw_pointer_cast(dev_results[num_layers-2].data()), 
              1, 
              &beta,thrust::raw_pointer_cast(&output[0]),
              1
            );
  if(which_fcn == SIGMOID)
    thrust::for_each(thrust::device,output.begin(),output.end(),logistic_functor());  
  else if(which_fcn == TANH)
    thrust::for_each(thrust::device,output.begin(),output.end(),tanh_functor());  

  (hipblasDestroy(handle));
  //hipStreamDestroy(Stream);  

}

void neural_network::compare(thrust::host_vector<float> &input, thrust::host_vector<float> &output)
{
  thrust::device_vector<float> original = input;
  thrust::device_vector<float> reconstructed = input;
  thrust::device_vector<float> descriptor;
 
  // float time;
  // hipEvent_t start, stop;

  // ( hipEventCreate(&start) );
  // ( hipEventCreate(&stop) );
  // ( hipEventRecord(start, 0) );

  encode(original,descriptor);

  // ( hipEventRecord(stop, 0) );
  // ( hipEventSynchronize(stop) );
  // ( hipEventElapsedTime(&time, start, stop) );
  //printf("%3.2f ", time);

  decode(descriptor,reconstructed);
  output = reconstructed;
}


void neural_network::compare_gold(thrust::host_vector<float> &input ,thrust::host_vector<float> &output )
{
  const int gold_num_layers = 10;
  int nodes[gold_num_layers+1] =     { 4096,     2048,     1024,     512 ,     256,      128,      256,      512,      1024,     2048,     4096};
  int num_outputs[gold_num_layers] = {           nodes[1], nodes[2], nodes[3], nodes[4], nodes[5], nodes[6], nodes[7], nodes[8], nodes[9], nodes[10] };
  int num_inputs[gold_num_layers] =  { nodes[0], nodes[1], nodes[2], nodes[3], nodes[4], nodes[5], nodes[6], nodes[7], nodes[8], nodes[9]};

  output.resize(nodes[gold_num_layers]);

  //allocating neural network container
  float* outputs[gold_num_layers];
  float* activations[gold_num_layers];

  for (int layer_number = 0; layer_number < gold_num_layers; ++layer_number)
  {
    outputs[layer_number] = new float[num_outputs[layer_number]];
    activations[layer_number] = new float[num_outputs[layer_number]];
  }

  //set activations and output to zero so we can compute the sums
  for(uint layer_number = 0; layer_number < gold_num_layers; layer_number++)
    for (uint fill = 0; fill < num_outputs[layer_number]; ++fill)
      outputs[layer_number][fill] = activations[layer_number][fill] = 0.f;

  for(uint layer_number = 0; layer_number < gold_num_layers; ++layer_number)
  {
    // #pragma omp parallel
    for(uint n_out = 0; n_out < num_outputs[layer_number]; ++n_out)
    {
      //compute activations layerwise
      for(uint n_in = 0; n_in < num_inputs[layer_number]; ++n_in)
      {

        activations[layer_number][n_out] += (layer_number == 0) ? (input[n_in]) /*-TRUNC_NEG)/(TRUNC_PLUS-TRUNC_NEG)*/ * host_weights[layer_number][ nodes[layer_number]*n_out + n_in ] :
                                            outputs[layer_number-1][n_in] * host_weights[layer_number][ nodes[layer_number]*n_out + n_in ];
      }
      outputs[layer_number][n_out] = logistic(activations[layer_number][n_out] + host_biases[layer_number][n_out] );
    }
  }

  for(int d=0; d < TSDF_BLOCKSIZE*TSDF_BLOCKSIZE*TSDF_BLOCKSIZE; ++d)
  {
    output[d] = outputs[gold_num_layers-1][d]/**(TRUNC_PLUS-TRUNC_NEG) + TRUNC_NEG*/;
  }

  for (int del = 0; del < gold_num_layers; ++del)
  {
    if(activations[del]!=NULL)
      delete[] activations[del];
    if(outputs[del]!=NULL)
      delete[] outputs[del];
  }
}
